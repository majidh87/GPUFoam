#include "hip/hip_runtime.h"
#include "discretizationKernel.h"
#define BSIZE 512
#define BSIZEX 32
#define BSIZEY 32

__global__ void cellKernel( double *vcs,
                            double* tot,
                            double rDelgaG,
                            double *diag, 
                            double *source,
                            int sizeDiag)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id>= sizeDiag)
        return;

    diag[id] = rDelgaG*vcs[id];
    source[id] = rDelgaG*vcs[id]*tot[id];
}

__global__ void faceKernel( double *delta,
                            double *gamma,
                            int *upperAddr,
                            int *lowerAddr,
                            double *upper, 
                            double *lower,
                            double *diag,
                            int sizeFace
                            )
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id>= sizeFace)
        return;

    double temp=delta[id]*gamma[id];
    lower[id] = temp;
    upper[id] = temp;
    atomicAdd(&diag[lowerAddr[id]], -temp);
    atomicAdd(&diag[upperAddr[id]], -temp);
}

__global__ void boundaryKernel( int *pSize,
                                int **pAdrr,
                                double **pf_BC,
                                double **pf_IC,
                                double **pf_GammaSf,
                                double *diag,
                                double *source,
                                int maxPatches,
                                int numberOfPatches
                                )
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;
    if (idx >= pSize[idy] || idy >= numberOfPatches)
		return;
    atomicAdd(&diag[pAdrr[idy][idx]],pf_GammaSf[idy][idx]*pf_IC[idy][idx]);
    atomicAdd(&source[pAdrr[idy][idx]],-pf_GammaSf[idy][idx]*pf_BC[idy][idx]);
    //printf("***** %f %f\n",source[pAdrr[idy][idx]],-pf_GammaSf[idy][idx]*pf_BC[idy][idx] );
}

__global__ void updateSourceKernel( double *vcs,
                            double* tot,
                            double rDelgaG,
                            double *source,
                            int sizeDiag)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id>= sizeDiag)
        return;

    source[id] = rDelgaG*vcs[id]*tot[id];
     //printf("***** %f %f\n",source[pAdrr[idy][idx]],-pf_GammaSf[idy][idx]*pf_BC[idy][idx] );
}

__global__ void updateBoundaryKernel( int *pSize,
                                int **pAdrr,
                                double **pf_BC,
                                double **pf_IC,
                                double **pf_GammaSf,
                                double *source,
                                int maxPatches,
                                int numberOfPatches
                                )
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;
    if (idx >= pSize[idy] || idy >= numberOfPatches)
		return;
    atomicAdd(&source[pAdrr[idy][idx]],-pf_GammaSf[idy][idx]*pf_BC[idy][idx]);
    //printf("***** %f %f\n",source[pAdrr[idy][idx]],-pf_GammaSf[idy][idx]*pf_BC[idy][idx] );
}

void discKernelWrapper( int sizeDiag,
                        int sizeFace,
                        double *vcs, 
                        double *tot,
                        double *delta,
                        double *gamma,
                        int *upperAddr,
                        int *lowerAddr,
                        int numOfPatches,
                        int maxPatches,
                        int *d_pSize,
                        int **d_pAdrr,
                        double **d_pf_BC,
                        double **d_pf_IC,
                        double **d_pf_GammaSf,
                        double rDelgaG,
                        double *d_diag,
                        double *d_source,
                        double *d_upper,
                        double *d_lower
                        )
{
    int blockSize = BSIZE;
    int gridCell = (int)ceil((float)(sizeDiag)/blockSize);
 
    cellKernel<<<gridCell, blockSize>>>(vcs,
                                        tot,
                                        rDelgaG,   
                                        d_diag, 
                                        d_source, 
                                        sizeDiag
                                        );

    
    int gridFace = (int)ceil((float)(sizeFace)/blockSize);
    faceKernel<<<gridFace, blockSize>>>(delta,
                                        gamma,  
                                        upperAddr,
                                        lowerAddr,
                                        d_upper,
                                        d_lower,
                                        d_diag,
                                        sizeFace
                                        );

    dim3 blockBoundary(BSIZEX, BSIZEY, 1);
	dim3 gridBoundary((int)ceil((float)(maxPatches)/blockBoundary.x), (int)ceil((float)(numOfPatches)/blockBoundary.y), 1);
    boundaryKernel<<<gridBoundary, blockBoundary>>>(d_pSize,
                                            d_pAdrr,
                                            d_pf_BC,
                                            d_pf_IC,
                                            d_pf_GammaSf,
                                            d_diag,
                                            d_source,
                                            maxPatches,
                                            numOfPatches
                                            );
   
    
}

void updateDiscWrapper(int sizeDiag,
                        double *vcs, 
                        double *tot,
                        int numOfPatches,
                        int maxPatches,
                        int *d_pSize,
                        int **d_pAdrr,
                        double **d_pf_BC,
                        double **d_pf_IC,
                        double **d_pf_GammaSf,
                        double rDelgaG,
                        double *d_source
                        )
{
    int blockSize = BSIZE;
    int gridCell = (int)ceil((float)(sizeDiag)/blockSize);
 
    updateSourceKernel<<<gridCell, blockSize>>>(vcs,
                                        tot,
                                        rDelgaG,    
                                        d_source, 
                                        sizeDiag
                                        );

    dim3 blockBoundary(BSIZEX, BSIZEY, 1);
	dim3 gridBoundary((int)ceil((float)(maxPatches)/blockBoundary.x), (int)ceil((float)(numOfPatches)/blockBoundary.y), 1);
    updateBoundaryKernel<<<gridBoundary, blockBoundary>>>(d_pSize,
                                            d_pAdrr,
                                            d_pf_BC,
                                            d_pf_IC,
                                            d_pf_GammaSf,
                                            d_source,
                                            maxPatches,
                                            numOfPatches
                                            );
   
    
}

